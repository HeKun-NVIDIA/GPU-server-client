#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define CHECK(res) if(res!=hipSuccess){exit(-1);}
__global__ void Kerneltest(char *da, char *dr)
{
  unsigned int i=  threadIdx.x;
  if (i <10)
  {
    dr[i] = da[i]+1;
  }
}
 
extern "C" int func(char *data, char *result) 
{
  char *da = NULL;
  char *ha = NULL;
  char *dr = NULL;
  char *hr = NULL;
  hipError_t res;
  int r;
  printf("2.GPU recieve :\n %s\n",data );
  res = hipMalloc((void**)(&da), 10*sizeof(char));CHECK(res)
  res = hipMalloc((void**)(&dr), 10*sizeof(char));CHECK(res)
  ha = (char*)malloc(10*sizeof(char));
  hr = (char*)malloc(10*sizeof(char));
 
  for (r = 0; r < 10; r++)
  {
    ha[r] = data[r];
  }
  res = hipMemcpy((void*)(da), (void*)(ha), 10*sizeof(char), hipMemcpyHostToDevice);CHECK(res)
  Kerneltest<<<1, 10>>>(da, dr);
  res = hipMemcpy((void*)(hr), (void*)(dr), 10*sizeof(char), hipMemcpyDeviceToHost);CHECK(res)
  memcpy(result,hr,10);
  printf("3.GPU func result:\n %s\n", result );
  hipFree((void*)da);
  hipFree((void*)dr);
  free(ha);
  free(hr);

  return 0;
}

