#include "hip/hip_runtime.h"

#include "comm.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void charAdd(char *a, char *c)
{
  int i = threadIdx.x;
  if(i<10)
  {
    printf("a[i]: %d\n",a[i]);
    c[i] = a[i]+1;
    printf("c[i]: %d\n",c[i]);
  }
}

int main()
{
    //char r[10];
    hipError_t err = hipSuccess;
    char *d_s = NULL;
    char *d_c = NULL;
    size_t size = 10*sizeof(char);
    int shmid=CreatShmid(4097);
    daemon(1,1);

    err=hipMalloc((void **)&d_s, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err=hipMalloc((void **)&d_c, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    if(shmid>0)
    {   
        //int i=0;
        char *addr=shmat(shmid,NULL,0);
        char data[10];
        while(20)
        {
          if (addr[0]=='a')
            {
              memcpy(data,addr,10);
              err = hipMemcpy(d_s, data, size, hipMemcpyHostToDevice);
              if (err != hipSuccess)
              {
                  fprintf(stderr, "Failed to copy string s from host to device (error code %s)!\n", hipGetErrorString(err));
                  exit(EXIT_FAILURE);
              }
              /*for(int i=0; i<10; i++){ 
                  data[i]=data[i]+1;
              }*/
              charAdd<<<1,10>>>(d_s, d_c);
              err = hipGetLastError();

              if (err != hipSuccess)
              {
                  fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
                  exit(EXIT_FAILURE);
              }
              err = hipMemcpy(data, d_c, size, hipMemcpyDeviceToHost);
              if (err != hipSuccess)
              {
                  fprintf(stderr, "Failed to copy vector d_s from device to host (error code %s)!\n", hipGetErrorString(err));
                  exit(EXIT_FAILURE);
              }
              hipFree(d_s);
              hipFree(d_c);
              memcpy(addr,data,10);
            }  
        }   
        if(shmdt(addr)==-1)
        {   
            perror("shmat");
            return -3; 
        }   
 
    }   
   else
    {   
        perror("CreatShmid");
        return -1;
    }
    if(Destory(shmid)<0)
    {
        perror("Destory");
        return -2;
    }
    return 0;
}
